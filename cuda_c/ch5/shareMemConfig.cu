#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    hipSharedMemConfig config;
    CHECK(hipDeviceGetSharedMemConfig(&config));
    printf("hipSharedMemConfig is %d\n", config);

    /*
    enum __device_builtin__ hipSharedMemConfig
    {
        hipSharedMemBankSizeDefault   = 0,
        hipSharedMemBankSizeFourByte  = 1,
        hipSharedMemBankSizeEightByte = 2
    };
    */

   hipFuncCache_t cacheConfig;
   hipDeviceSetCacheConfig(cacheConfig);
   /*
    enum __device_builtin__ hipFuncCache_t
    {
        hipFuncCachePreferNone   = 0,    // < Default function cache configuration, no preference
        hipFuncCachePreferShared = 1,    //**< Prefer larger shared memory and smaller L1 cache
        hipFuncCachePreferL1     = 2,    //**< Prefer larger L1 cache and smaller shared memory
        hipFuncCachePreferEqual  = 3     //**< Prefer equal size L1 cache and shared memory
    };
   */

    return 0;
}