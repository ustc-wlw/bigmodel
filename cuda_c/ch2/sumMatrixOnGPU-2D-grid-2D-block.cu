#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define CHECK(call)                                                              \
{                                                                                \
    const hipError_t error = call;                                              \
    if (error != hipSuccess) {                                                  \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                            \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));      \
    }                                                                            \
}                                                                                \

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);

    return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    int match = 1;
    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Array does not match\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
        
    }
    if (match)
    {
        printf("Array match success!!\n");
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int nx, const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy<ny; iy++){
        for (size_t ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx;
        ib += nx;
        ic += nx;
    }
}

__global__ void sumArraysOnDevice(float *A, float *B, float *C, const int nx, const int ny) {
    unsigned int ix = blockIdx.x * blockDim.x +  threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny) C[idx] = A[idx] + B[idx];
}

void initData(float *ip, int size) {
    time_t t;
    srand((unsigned int) time(&t));

    for (size_t i = 0; i < size; i++)
    {
        ip[i] = (float) (rand() & 0xFF) / 10.0f;
    }
}

int main() {
    printf("main start ............\n");

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("device propertites: %s\n", deviceProp.name);
    CHECK(hipSetDevice(dev));

    int nx = 1 << 14;
    int ny = 1 << 14;
    int n_Elem = nx * ny;
    printf("Matrix size %d, %d\n", nx, ny);
    size_t nBytes = n_Elem * sizeof(float);

    float *h_A, *h_B, *h_C, *gpuRef;
    h_A = (float*) malloc(nBytes);
    h_B = (float*) malloc(nBytes);
    h_C = (float*) malloc(nBytes);
    gpuRef = (float*) malloc(nBytes);

    double iStart, iElaps;

    printf("init host data start ....\n");
    iStart = cpuSecond();
    initData(h_A, n_Elem);
    initData(h_B, n_Elem);
    iElaps = cpuSecond() - iStart;
    printf("init cpu data time elaps: %5.2f\n", iElaps);

    memset(h_C, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);


    printf("After init data and copy data from host to device start....\n");
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    printf("copy data from host to device finished!!\n");

    int dimx = 32;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    iStart = cpuSecond();
    sumArraysOnDevice<<<grid, block>>> (d_A, d_B, d_C, nx, ny);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("sumArraysOnDevice <<<(%d, %d), (%d, %d)>>> time elaps: %f seconds\n", grid.x, grid.y, block.x, block.y, iElaps);

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    printf("kernel result has copy to host\n");

    iStart = cpuSecond();
    sumArraysOnHost(h_A, h_B, h_C, nx, ny);
    iElaps = cpuSecond() - iStart;
    printf("sumArraysOnHost time elaps: %f second\n", iElaps);
    
    printf("begain check result between host and device result...\n");
    checkResult(h_C, gpuRef, n_Elem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    free(h_A);
    free(h_B);
    free(h_C);
    free(gpuRef);

}