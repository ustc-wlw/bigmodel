#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define CHECK(call)                                                              \
{                                                                                \
    const hipError_t error = call;                                              \
    if (error != hipSuccess) {                                                  \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                            \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));      \
    }                                                                            \
}                                                                                \

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);

    return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    int match = 1;
    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Array does not match\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
        
    }
    if (match)
    {
        printf("Array match success!!\n");
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int i = 0; i<N; i++){
        C[i] = A[i] + B[i];
    }
}

__global__ void sumArraysOnDevice(float *A, float *B, float *C, const int N) {
    int tid = blockIdx.x * blockDim.x +  threadIdx.x;
    if (tid < N) C[tid] = A[tid] + B[tid];
}

void initData(float *ip, int size) {
    time_t t;
    srand((unsigned int) time(&t));

    for (size_t i = 0; i < size; i++)
    {
        ip[i] = (float) (rand() & 0xFF) / 10.0f;
    }
}

int main() {
    printf("main start ............\n");

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("device propertites: %s\n", deviceProp.name);
    CHECK(hipSetDevice(dev));

    int n_Elem = 1 << 24;
    printf("vector size %d\n", n_Elem);
    size_t nBytes = n_Elem * sizeof(float);

    float *h_A, *h_B, *h_C, *gpuRef;
    h_A = (float*) malloc(nBytes);
    h_B = (float*) malloc(nBytes);
    h_C = (float*) malloc(nBytes);
    gpuRef = (float*) malloc(nBytes);

    double iStart, iElaps;

    printf("init host data start ....\n");
    iStart = cpuSecond();
    initData(h_A, n_Elem);
    initData(h_B, n_Elem);
    iElaps = cpuSecond() - iStart;
    printf("init cpu data time elaps: %5.2f\n", iElaps);

    memset(h_C, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);


    printf("After init data and copy data from host to device start....\n");
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    printf("copy data from host to device finished!!\n");

    dim3 block(256);
    dim3 grid((n_Elem + block.x - 1) / block.x);

    iStart = cpuSecond();
    sumArraysOnDevice<<<grid, block>>> (d_A, d_B, d_C, n_Elem);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("sumArraysOnDevice <<<%d, %d>>> time elaps: %f seconds\n", grid.x, block.x, iElaps);

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    printf("kernel result has copy to host\n");

    iStart = cpuSecond();
    sumArraysOnHost(h_A, h_B, h_C, n_Elem);
    iElaps = cpuSecond() - iStart;
    printf("sumArraysOnHost time elaps: %f second\n", iElaps);
    
    printf("begain check result between host and device result...\n");
    checkResult(h_C, gpuRef, n_Elem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    free(h_A);
    free(h_B);
    free(h_C);
    free(gpuRef);

}