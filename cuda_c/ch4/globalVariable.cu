#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using a statically declared global variable (devData) to store
 * a floating-point value on the device.
 */

__device__ float devData;

__global__ void checkGlobalVariable()
{
    printf("Device: original device global variabel is %f\n", devData);

    devData += 2.0;
}

int main()
{
    float value = 3.14;
    // CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));
    float *devPtr;
    CHECK(hipGetSymbolAddress((void**)&devPtr, devData));
    CHECK(hipMemcpy(devPtr, &value, sizeof(float), hipMemcpyHostToDevice));
    printf("Host: copy value %f to device\n", value);

    checkGlobalVariable<<<1, 1>>> ();
    // CHECK(hipDeviceSynchronize());

    // CHECK(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
    CHECK(hipMemcpy(&value, devPtr, sizeof(float), hipMemcpyDeviceToHost));
    printf("Host: after kenerl run, value is %f\n", value);

    CHECK(hipDeviceReset());
    return 0;

}