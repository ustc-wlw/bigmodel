#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

inline void enableP2P(const int ngpus)
{
    for (int i = 0; i < ngpus; i++)
    {
        hipSetDevice(i);
        for (int j = 0; j < ngpus; j++)
        {
            if (i == j) continue;
            int peer_device_available = 0;
            hipDeviceCanAccessPeer(&peer_device_available, i, j);
            if (peer_device_available)
            {
                hipDeviceEnablePeerAccess(j, 0);
                printf("GPU%d,  can access peer memory of GPU%d \n", i, j);
            }
            else
            {
                printf("(GPU%d, GPU%d) can not access peer memory\n", i, j);
            }
        }
    }
}

inline void disableP2P (int ngpus)
{
    for( int i = 0; i < ngpus; i++ )
    {
        CHECK(hipSetDevice(i));

        for(int j = 0; j < ngpus; j++)
        {
            if( i == j ) continue;

            int peer_access_available = 0;
            CHECK(hipDeviceCanAccessPeer( &peer_access_available, i, j) );

            if( peer_access_available )
            {
                CHECK(hipDeviceDisablePeerAccess(j));
                printf("> GPU%d disabled direct access to GPU%d\n", i, j);
            }
        }
    }
}

int main(int args, char **argv)
{
    int ngpus;
    CHECK(hipGetDeviceCount(&ngpus));
    printf(" CUDA capable devices: %i\n", ngpus);

    // enableP2P(ngpus);

    int nElem = 1 << 10;
    int nBytes = nElem * sizeof(float);

    float *d_mem[2];
    for (size_t i = 0; i < 2; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipMalloc((void **)&d_mem[i], nBytes));
        CHECK(hipMemset(d_mem[i], i, nElem));
    }
    
    CHECK(hipSetDevice(0));
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    hipEventRecord(start);
    for (size_t i = 0; i < 100; i++)
    {
        if (i % 2 == 0)
        {
            CHECK(hipMemcpy(d_mem[1], d_mem[0], nBytes, hipMemcpyDeviceToDevice));
        } else {
            CHECK(hipMemcpy(d_mem[0], d_mem[1],  nBytes, hipMemcpyDeviceToDevice));
        }
    }
    
    CHECK(hipSetDevice(0));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time_ms;
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    elapsed_time_ms /= 100.0f;
    printf("Ping-pong unidirectional hipMemcpy:\t\t %8.2f ms ",
           elapsed_time_ms);
    printf("performance: %8.2f GB/s\n",
            (float)nBytes / (elapsed_time_ms * 1e6f));

    for (size_t i = 0; i < 2; i++)
    {
        CHECK(hipSetDevice(i));
        CHECK(hipFree(d_mem[i]));
    }

    CHECK(hipSetDevice(0));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

}

