#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void naiveCopy(int *in, int *o, const int N)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N)
    {
        o[idx] = in[idx];
    }
}

__global__ void kenerl1()
{
    printf("kernal 1 run .....\n");
    double sum = 0.0;
    for(int i = 0; i< 100000000; i++)
    {
        sum += i;
    }
    printf("kernal 1 finished !!\n");
}

__global__ void kenerl2()
{
    printf("kernal 2 run .....\n");
    double sum = 0.0;
    for(int i = 0; i< 100000000; i++)
    {
        sum += i;
    }
    printf("kernal 2 finished !!!\n");
}

__global__ void kenerl3()
{
    printf("kernal 3 run .....\n");
    double sum = 0.0;
    for(int i = 0; i< 100000000; i++)
    {
        sum += i;
    }
    printf("kernal 3 finished !!!!!\n");
}

int main()
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));


    hipStream_t pStream1, pStream2;
    // CHECK(hipStreamCreate(&pStream1));
    // CHECK(hipStreamCreate(&pStream2));
    CHECK(hipStreamCreateWithFlags(&pStream1, hipStreamNonBlocking));
    CHECK(hipStreamCreateWithFlags(&pStream2, hipStreamNonBlocking));

    dim3 block(1);
    dim3 grid(1);

    kenerl1<<<grid, block, 0, pStream1>>> ();
    kenerl2<<<grid, block>>> ();
    kenerl3<<<grid, block, 0, pStream2>>> ();

    hipDeviceSynchronize();

    // hipStreamSynchronize(pStream);
    CHECK(hipStreamDestroy(pStream1));
    CHECK(hipStreamDestroy(pStream2));

    return 0;
}