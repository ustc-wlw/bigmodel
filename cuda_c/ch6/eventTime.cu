#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernal(int *in, int *o, const int N)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N)
    {
        o[idx] = in[idx];
    }
}

int main()
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    int nElem = 1 << 14;
    size_t nBytes = nElem * sizeof(int);

    dim3 block(32);
    dim3 grid(nElem / block.x);

    int *d_a;
    int *d_b;
    hipMalloc((void**)&d_a, nBytes);
    hipMemset((void *)d_a, 1, nElem);

    hipMalloc((void**)&d_b, nBytes);
    hipMemset((void *)d_b, 0, nElem);

    hipStream_t pStream;
    CHECK(hipStreamCreate(&pStream));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    hipEventRecord(start, pStream);

    kernal<<<grid, block, 0, pStream>>>(d_a, d_b, nElem);

    hipEventRecord(stop, pStream);

    hipEventSynchronize(stop);

    float elapsed_time = 0.0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("kernal elapsed time is %f\n", elapsed_time);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    // hipStreamSynchronize(pStream);
    CHECK(hipStreamDestroy(pStream));
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));

    return 0;
}