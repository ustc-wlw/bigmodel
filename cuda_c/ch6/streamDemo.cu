#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    int leastPriority;
    int greatestPriority;
    CHECK(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));
    printf("least Priority is %d, greatest Priority is %d\n", leastPriority, greatestPriority);

    hipStream_t pStream;
    CHECK(hipStreamCreate(&pStream));
    CHECK(hipStreamDestroy(pStream));

    hipEvent_t pEvent;
    CHECK(hipEventCreate(&pEvent));
    CHECK(hipEventDestroy(pEvent));



    return 0;
}